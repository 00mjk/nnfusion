
#include <hip/hip_runtime.h>
extern "C" __global__ void manual_dot_nn_op_float_m1_k256_n256_kernel0(float* input0, float* input1, float* output0)
{
    int warp_id = threadIdx.x >> 5;
    int lane_id = threadIdx.x & 31;
    int col_id = blockIdx.x * blockDim.x / 4 + lane_id;
    if (col_id < 256)
    {
        float val = 0;
        int k_start = warp_id * 64;
        int k_end = (warp_id + 1) * 64;
        for (int i = k_start; i < k_end; i++)
        {
            val = fma(input0[i], input1[i * 256 + col_id], val);
        }
        if (warp_id == 0)
        {
            output0[col_id]=0;
        }
        __syncthreads();
        atomicAdd(output0 + col_id, val);
    }

}
